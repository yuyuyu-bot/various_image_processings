#include "hip/hip_runtime.h"
#include "bilateral_filter_impl.cuh"
#include "device_utilities.cuh"
#include "host_utilities.hpp"

constexpr auto kernel_color_table_len = 256 * 3;

__global__ void bilateral_filter_kernel(
    const std::uint8_t* const src,
    std::uint8_t* const       dst,
    const int                 ksize,
    const float* const        kernel_space,
    const float* const        kernel_color_table,
    const int                 width,
    const int                 height
) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int stride_3ch = width * 3;
    const int radius  = ksize / 2;

    extern __shared__ float s_bilateral_filter_buffer[];
    auto s_kernel_space       = &s_bilateral_filter_buffer[0];
    auto s_kernel_color_table = &s_bilateral_filter_buffer[ksize * ksize];

    const int smem_width    = blockDim.x + ksize - 1;
    const int smem_height   = blockDim.y + ksize - 1;
    const int smem_stride   = smem_width * 3;
    const int smem_origin_x = x - tx - radius;
    const int smem_origin_y = y - ty - radius;
    auto s_src              = reinterpret_cast<std::uint8_t*>(&s_kernel_color_table[kernel_color_table_len]);

    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < ksize * ksize; i += blockDim.x * blockDim.y) {
        s_kernel_space[i] = kernel_space[i];
    }
    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < kernel_color_table_len; i += blockDim.x * blockDim.y) {
        s_kernel_color_table[i] = kernel_color_table[i];
    }

    const auto get_s_src_ptr = [s_src, smem_stride, smem_origin_x, smem_origin_y](const int src_x, const int src_y) {
        const auto s_src_x = src_x - smem_origin_x;
        const auto s_src_y = src_y - smem_origin_y;
        return &s_src[smem_stride * s_src_y + s_src_x * 3];
    };

    for (int y_offset = ty; y_offset < smem_height; y_offset += blockDim.y) {
        for (int x_offset = tx; x_offset < smem_width; x_offset += blockDim.x) {
            auto* const s_src_ptr = get_s_src_ptr(smem_origin_x + x_offset, smem_origin_y + y_offset);
            const auto x_clamped = clamp(smem_origin_x + x_offset, 0, width - 1);
            const auto y_clamped = clamp(smem_origin_y + y_offset, 0, height - 1);
            s_src_ptr[0] = src[stride_3ch * y_clamped + x_clamped * 3 + 0];
            s_src_ptr[1] = src[stride_3ch * y_clamped + x_clamped * 3 + 1];
            s_src_ptr[2] = src[stride_3ch * y_clamped + x_clamped * 3 + 2];
        }
    }
    __syncthreads();

    if (x >= width || y >= height) {
        return;
    }

    const auto get_kernel_space = [ksize, radius, s_kernel_space](const int kx, const int ky) {
        return s_kernel_space[(ky + radius) * ksize + (kx + radius)];
    };

    const auto get_kernel_color = [s_kernel_color_table](const auto a, const auto b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = abs(diff0) + abs(diff1) + abs(diff2);
        return s_kernel_color_table[color_distance];
    };

    const auto src_center_pix = src + stride_3ch * y + x * 3;
    auto sum0 = 0.f;
    auto sum1 = 0.f;
    auto sum2 = 0.f;
    auto sumk = 0.f;

    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto src_pix   = get_s_src_ptr(x + kx, y + ky);
            const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(src_center_pix, src_pix);

            sum0 += src_pix[0] * kernel;
            sum1 += src_pix[1] * kernel;
            sum2 += src_pix[2] * kernel;
            sumk += kernel;
        }
    }

    dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk + 0.5f);
    dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk + 0.5f);
    dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk + 0.5f);
}

__global__ void joint_bilateral_filter_kernel(
    const std::uint8_t* const src,
    const std::uint8_t* const guide,
    std::uint8_t* const       dst,
    const int                 ksize,
    const float* const        kernel_space,
    const float* const        kernel_color_table,
    const int                 width,
    const int                 height
) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int stride_3ch = width * 3;
    const int radius  = ksize / 2;

    extern __shared__ float s_joint_bilateral_filter_buffer[];
    auto s_kernel_space       = &s_joint_bilateral_filter_buffer[0];
    auto s_kernel_color_table = &s_joint_bilateral_filter_buffer[ksize * ksize];

    const int smem_width    = blockDim.x + ksize - 1;
    const int smem_height   = blockDim.y + ksize - 1;
    const int smem_stride = smem_width * 3;
    const int smem_origin_x = x - tx - radius;
    const int smem_origin_y = y - ty - radius;
    auto s_src              = reinterpret_cast<std::uint8_t*>(&s_kernel_color_table[kernel_color_table_len]);
    auto s_guide            = &s_src[smem_stride * smem_height];

    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < ksize * ksize; i += blockDim.x * blockDim.y) {
        s_kernel_space[i] = kernel_space[i];
    }
    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < kernel_color_table_len; i += blockDim.x * blockDim.y) {
        s_kernel_color_table[i] = kernel_color_table[i];
    }

    const auto get_s_src_ptr = [s_src, smem_stride, smem_origin_x, smem_origin_y](const int src_x, const int src_y) {
        const auto s_src_x = src_x - smem_origin_x;
        const auto s_src_y = src_y - smem_origin_y;
        return &s_src[smem_stride * s_src_y + s_src_x * 3];
    };

    const auto get_s_guide_ptr = [s_guide, smem_stride, smem_origin_x, smem_origin_y](const int guide_x, const int guide_y) {
        const auto s_guide_x = guide_x - smem_origin_x;
        const auto s_guide_y = guide_y - smem_origin_y;
        return &s_guide[smem_stride * s_guide_y + s_guide_x * 3];
    };

    for (int y_offset = ty; y_offset < smem_height; y_offset += blockDim.y) {
        for (int x_offset = tx; x_offset < smem_width; x_offset += blockDim.x) {
            auto* const s_src_ptr   = get_s_src_ptr(smem_origin_x + x_offset, smem_origin_y + y_offset);
            auto* const s_guide_ptr = get_s_guide_ptr(smem_origin_x + x_offset, smem_origin_y + y_offset);
            const auto x_clamped    = clamp(smem_origin_x + x_offset, 0, width - 1);
            const auto y_clamped    = clamp(smem_origin_y + y_offset, 0, height - 1);
            s_src_ptr[0]   = src[stride_3ch * y_clamped + x_clamped * 3 + 0];
            s_src_ptr[1]   = src[stride_3ch * y_clamped + x_clamped * 3 + 1];
            s_src_ptr[2]   = src[stride_3ch * y_clamped + x_clamped * 3 + 2];
            s_guide_ptr[0] = guide[stride_3ch * y_clamped + x_clamped * 3 + 0];
            s_guide_ptr[1] = guide[stride_3ch * y_clamped + x_clamped * 3 + 1];
            s_guide_ptr[2] = guide[stride_3ch * y_clamped + x_clamped * 3 + 2];
        }
    }
    __syncthreads();

    if (x >= width || y >= height) {
        return;
    }

    const auto get_kernel_space = [ksize, radius, s_kernel_space](const int kx, const int ky) {
        return s_kernel_space[(ky + radius) * ksize + (kx + radius)];
    };

    const auto get_kernel_color = [s_kernel_color_table](const auto a, const auto b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = abs(diff0) + abs(diff1) + abs(diff2);
        return s_kernel_color_table[color_distance];
    };

    const auto guide_center_pix = guide + stride_3ch * y + x * 3;
    auto sum0 = 0.f;
    auto sum1 = 0.f;
    auto sum2 = 0.f;
    auto sumk = 0.f;

    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);
            const auto src_pix   = get_s_src_ptr(x + kx, y + ky);
            const auto guide_pix = get_s_guide_ptr(x + kx, y + ky);
            const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(guide_center_pix, guide_pix);

            sum0 += src_pix[0] * kernel;
            sum1 += src_pix[1] * kernel;
            sum2 += src_pix[2] * kernel;
            sumk += kernel;
        }
    }

    dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk + 0.5f);
    dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk + 0.5f);
    dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk + 0.5f);
}

CudaBilateralFilter::Impl::Impl(
    const int   width,
    const int   height,
    const int   ksize,
    const float sigma_space,
    const float sigma_color)
: width_(width),
  height_(height),
  ksize_(ksize),
  sigma_space_(sigma_space),
  sigma_color_(sigma_color),
  d_kernel_space_(ksize * ksize),
  d_kernel_color_table_(kernel_color_table_len) {
    const auto gauss_color_coeff = -1.f / (2 * sigma_color * sigma_color);
    const auto gauss_space_coeff = -1.f / (2 * sigma_space * sigma_space);
    const auto radius  = ksize / 2;

    std::vector<float> h_kernel_space(ksize_ * ksize_);
    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto kidx = (ky + radius) * ksize_ + (kx + radius);
            const auto r2 = kx * kx + ky * ky;
            if (r2 > radius * radius) {
                continue;
            }
            h_kernel_space[kidx] = std::exp(r2 * gauss_space_coeff);
        }
    }
    thrust::copy(h_kernel_space.begin(), h_kernel_space.end(), d_kernel_space_.begin());

    std::vector<float> h_kernel_color_table(kernel_color_table_len);
    for (int i = 0; i < h_kernel_color_table.size(); i++) {
        h_kernel_color_table[i] = std::exp((i * i) * gauss_color_coeff);
    }
    thrust::copy(h_kernel_color_table.begin(), h_kernel_color_table.end(), d_kernel_color_table_.begin());
}

void CudaBilateralFilter::Impl::bilateral_filter(
    const std::uint8_t* const d_src,
    std::uint8_t* const       d_dst
) const {
    const std::uint32_t block_width  = 32u;
    const std::uint32_t block_height = 32u;
    const std::uint32_t grid_width   = (width_  + block_width  - 1) / block_width;
    const std::uint32_t grid_height  = (height_ + block_height - 1) / block_height;

    const dim3 grid_dim (grid_width, grid_height);
    const dim3 block_dim(block_width, block_height);
    const std::uint32_t smem_size =
        (d_kernel_space_.size() + d_kernel_color_table_.size()) * sizeof(float) +
        (block_width + ksize_ - 1) * (block_height + ksize_ - 1) * 3 * sizeof(std::uint8_t);
    bilateral_filter_kernel<<<grid_dim, block_dim, smem_size>>>(
        d_src, d_dst, ksize_, d_kernel_space_.data().get(), d_kernel_color_table_.data().get(), width_, height_);
    CUDASafeCall();
}

void CudaBilateralFilter::Impl::joint_bilateral_filter(
    const std::uint8_t* const d_src,
    const std::uint8_t* const d_guide,
    std::uint8_t* const       d_dst
) const {
    const std::uint32_t block_width  = 32u;
    const std::uint32_t block_height = 32u;
    const std::uint32_t grid_width   = (width_  + block_width  - 1) / block_width;
    const std::uint32_t grid_height  = (height_ + block_height - 1) / block_height;

    const dim3 grid_dim (grid_width, grid_height);
    const dim3 block_dim(block_width, block_height);
    const std::uint32_t smem_size =
        (d_kernel_space_.size() + d_kernel_color_table_.size()) * sizeof(float) +
        (block_width + ksize_ - 1) * (block_height + ksize_ - 1) * 3 * sizeof(std::uint8_t) +
        (block_width + ksize_ - 1) * (block_height + ksize_ - 1) * 3 * sizeof(std::uint8_t);
    joint_bilateral_filter_kernel<<<grid_dim, block_dim, smem_size>>>(
        d_src, d_guide, d_dst, ksize_, d_kernel_space_.data().get(), d_kernel_color_table_.data().get(),
        width_, height_);
    CUDASafeCall();
}

CudaBilateralFilter::CudaBilateralFilter(
    const int   width,
    const int   height,
    const int   ksize,
    const float sigma_space,
    const float sigma_color
) {
    impl_ = new CudaBilateralFilter::Impl(width, height, ksize, sigma_space, sigma_color);
}

CudaBilateralFilter::~CudaBilateralFilter() {
    delete impl_;
}

void CudaBilateralFilter::bilateral_filter(
    const std::uint8_t* const d_src,
    std::uint8_t* const d_dst
) const {
    impl_->bilateral_filter(d_src, d_dst);
    hipDeviceSynchronize();
}


void CudaBilateralFilter::joint_bilateral_filter(
    const std::uint8_t* const d_src,
    const std::uint8_t* const d_guide,
    std::uint8_t* const d_dst
) const {
    impl_->joint_bilateral_filter(d_src, d_guide, d_dst);
    hipDeviceSynchronize();
}
