#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "bilateral_texture_filter.hpp"



#define CUDASafeCall() cuda_safe_call(hipGetLastError(), __FILE__, __LINE__);

static constexpr auto epsilon = 1e-9;

inline void cuda_safe_call(const hipError_t& error, const char* const file, const int line) {
    if (error != hipSuccess) {
        std::fprintf(stderr, "CUDA Error %s : %d %s\n", file, line, hipGetErrorString(error));
    }
}

template <typename T>
inline __device__ T clamp(T v, T min, T max) {
    return v < min ? min :
           v > max ? max :
           v;
}

template <typename ImageType, typename MagnitudeType>
__global__ void compute_magnitude_kernel(const ImageType* const image, MagnitudeType* const magnitude,
                                         const int width, const int height) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride_3ch = width * 3;
    const int stride = width;
    const int x = idx % width;
    const int y = idx / width;

    if (x == 0 || x == width - 1 || y == 0 || y == height - 1) {
        magnitude[stride * y + x] = 0.f;
        return;
    }

    const auto compute_del = [image, stride_3ch](const int x0, const int y0, const int x1, const int y1) {
        const auto diff0 = image[stride_3ch * y0 + x0 * 3 + 0] - image[stride_3ch * y1 + x1 * 3 + 0];
        const auto diff1 = image[stride_3ch * y0 + x0 * 3 + 1] - image[stride_3ch * y1 + x1 * 3 + 1];
        const auto diff2 = image[stride_3ch * y0 + x0 * 3 + 2] - image[stride_3ch * y1 + x1 * 3 + 2];
        return diff0 * diff0 + diff1 * diff1 + diff2 * diff2;
    };

    const auto del_x = compute_del(x - 1, y, x + 1, y);
    const auto del_y = compute_del(x, y - 1, x, y + 1);
    magnitude[stride * y + x] = sqrtf(del_x + del_y);
}

template <typename ImageType, typename MagnitudeType, typename BlurredType, typename RTVType>
__global__ void compute_blur_and_rtv_kernel(const ImageType* const image, const MagnitudeType* const magnitude,
                                            BlurredType* const blurred, RTVType* const rtv, const int ksize,
                                            const int width, const int height) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride_3ch = width * 3;
    const int stride = width;
    const int x = idx % width;
    const int y = idx / width;
    const int khalf  = ksize / 2;

    const auto get_intensity = [image, stride_3ch](const int x0, const int y0) {
        const auto pix = image + stride_3ch * y0 + x0 * 3;
        return (pix[0] + pix[1] + pix[2]) / 3.f;
    };

    auto sum0 = 0;
    auto sum1 = 0;
    auto sum2 = 0;

    auto intensity_max = 0.f;
    auto intensity_min = 0.f;
    auto magnitude_max      = 0.f;
    auto magnitude_sum      = 0.f;

    for (int ky = -khalf; ky <= khalf; ky++) {
        for (int kx = -khalf; kx <= khalf; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);

            sum0 += image[stride_3ch * y_clamped + x_clamped * 3 + 0];
            sum1 += image[stride_3ch * y_clamped + x_clamped * 3 + 1];
            sum2 += image[stride_3ch * y_clamped + x_clamped * 3 + 2];

            intensity_max  = max(intensity_max, get_intensity(x_clamped, y_clamped));
            intensity_min  = min(intensity_min, get_intensity(x_clamped, y_clamped));
            magnitude_max  = max(magnitude_max, magnitude[stride * y_clamped + x_clamped]);
            magnitude_sum += magnitude[stride * y_clamped + x_clamped];
        }
    }

    blurred[stride_3ch * y + x * 3 + 0] = static_cast<BlurredType>(sum0 / (ksize * ksize));
    blurred[stride_3ch * y + x * 3 + 1] = static_cast<BlurredType>(sum1 / (ksize * ksize));
    blurred[stride_3ch * y + x * 3 + 2] = static_cast<BlurredType>(sum2 / (ksize * ksize));
    rtv[stride * y + x] = (intensity_max - intensity_min) * magnitude_max / (magnitude_sum + epsilon);
}

template <typename BlurredType, typename RTVType, typename GuideType>
__global__ void compute_guide_kernel(const BlurredType* const blurred, const RTVType* const rtv, GuideType* const guide,
                                     const int ksize, const int width, const int height) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride_3ch = width * 3;
    const int stride = width;
    const int x = idx % width;
    const int y = idx / width;
    const auto khalf  = ksize / 2;
    const auto sigma_alpha = 1.f / (5 * ksize);

    auto rtv_min = 1e10f;
    auto rtv_min_x = 0;
    auto rtv_min_y = 0;

    for (int ky = -khalf; ky <= khalf; ky++) {
        for (int kx = -khalf; kx <= khalf; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);

            if (rtv_min > rtv[stride * y_clamped + x_clamped]) {
                rtv_min = rtv[stride * y_clamped + x_clamped];
                rtv_min_x = x_clamped;
                rtv_min_y = y_clamped;
            }
        }
    }

    const auto alpha =
        2 / (1 + exp(sigma_alpha * (rtv[stride * y + x] - rtv[stride * rtv_min_y + rtv_min_x]))) - 1.f;
    guide[stride_3ch * y + x * 3 + 0] =      alpha  * blurred[stride_3ch * rtv_min_y + rtv_min_x * 3 + 0] +
                                        (1 - alpha) * blurred[stride_3ch * y + x * 3 + 0];
    guide[stride_3ch * y + x * 3 + 1] =      alpha  * blurred[stride_3ch * rtv_min_y + rtv_min_x * 3 + 1] +
                                        (1 - alpha) * blurred[stride_3ch * y + x * 3 + 1];
    guide[stride_3ch * y + x * 3 + 2] =      alpha  * blurred[stride_3ch * rtv_min_y + rtv_min_x * 3 + 2] +
                                        (1 - alpha) * blurred[stride_3ch * y + x * 3 + 2];
}

template <typename ImageType, typename GuideType>
__global__ void joint_bilateral_filter_kernel(const ImageType* const src, const GuideType* const guide,
                                              ImageType* const dst, const int ksize, const float* const kernel_space,
                                              const float* const kernel_color_table, const int width,
                                              const int height) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride_3ch = width * 3;
    const int x = idx % width;
    const int y = idx / width;
    const auto khalf  = ksize / 2;

    const auto get_kernel_space = [ksize, khalf, kernel_space](const int kx, const int ky) {
        return kernel_space[(ky + khalf) * ksize + (kx + khalf)];
    };

    const auto get_kernel_color = [kernel_color_table](const auto a, const auto b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = (diff0 * diff0 + diff1 * diff1 + diff2 * diff2) / 3;
        return kernel_color_table[color_distance];
    };

    const auto guide_center_pix = guide + stride_3ch * y + x * 3;
    auto sum0 = 0.f;
    auto sum1 = 0.f;
    auto sum2 = 0.f;
    auto sum_k = 0.f;

    for (int ky = -khalf; ky <= khalf; ky++) {
        for (int kx = -khalf; kx <= khalf; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);
            const auto pix       = src + stride_3ch * y_clamped + x_clamped * 3;
            const auto guide_pix = guide + stride_3ch * y_clamped + x_clamped * 3;
            const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(guide_center_pix, guide_pix);

            sum0 += pix[0] * kernel;
            sum1 += pix[1] * kernel;
            sum2 += pix[2] * kernel;
            sum_k += kernel;
        }
    }

    dst[stride_3ch * y + x * 3 + 0] = static_cast<ImageType>(sum0 / sum_k);
    dst[stride_3ch * y + x * 3 + 1] = static_cast<ImageType>(sum1 / sum_k);
    dst[stride_3ch * y + x * 3 + 2] = static_cast<ImageType>(sum2 / sum_k);
}

#include "debug_show.hpp"
class CudaBilateralTextureFilterImpl {
public:
    CudaBilateralTextureFilterImpl(const int width, const int height) : width_(width), height_(height) {
        d_src_n_     = thrust::device_vector<std::uint8_t>(width_ * height_ * 3);
        d_blurred_   = thrust::device_vector<float>(width_ * height_ * 3);
        d_magnitude_ = thrust::device_vector<float>(width_ * height_);
        d_rtv_       = thrust::device_vector<float>(width_ * height_);
        d_guide_     = thrust::device_vector<std::uint8_t>(width_ * height_ * 3);
    }

    ~CudaBilateralTextureFilterImpl() {}

    template <typename ImageType>
    void execute(const thrust::device_vector<ImageType>& d_src, thrust::device_vector<ImageType>& d_dst,
                 const int ksize, const int nitr = 3, const bool debug_print = false) {
        thrust::copy(d_src.begin(), d_src.end(), d_dst.begin());

        for (int itr = 0; itr < nitr; itr++) {
            if (debug_print) { std::cout << cv::format("itration %d", itr + 1) << std::endl; }

            thrust::copy(d_dst.begin(), d_dst.end(), d_src_n_.begin());

            if (debug_print) { std::cout << "\tcompute magnitude ..." << std::endl; }
            compute_magnitude(d_src_n_, d_magnitude_);

            if (debug_print) { std::cout << "\tcompute rtv ..." << std::endl; }
            compute_blur_and_rtv(d_src_n_, d_magnitude_, d_blurred_, d_rtv_, ksize);

            if (debug_print) { std::cout << "\tcompute guide ..." << std::endl; }
            compute_guide(d_blurred_, d_rtv_, d_guide_, ksize);

            if (debug_print) { std::cout << "\tapply joint bilateral filter ..." << std::endl; }
            joint_bilateral_filter(d_src_n_, d_guide_, d_dst, 2 * ksize - 1, ksize - 1, jbf_sigma_color);
        }

        hipDeviceSynchronize();
    }


private:
    using ElemType = std::uint8_t;

    template <typename ImageType, typename MagnitudeType>
    void compute_magnitude(const thrust::device_vector<ImageType>& d_image,
                           thrust::device_vector<MagnitudeType>& d_magnitude) {
        const dim3 grid_dim{static_cast<std::uint32_t>(height_)};
        const dim3 block_dim{static_cast<std::uint32_t>(width_)};
        compute_magnitude_kernel<<<grid_dim, block_dim>>>(
            d_image.data().get(), d_magnitude.data().get(), width_, height_);
        CUDASafeCall();
    }

    template <typename ImageType, typename MagnitudeType, typename BlurredType, typename RTVType>
    void compute_blur_and_rtv(const thrust::device_vector<ImageType>& d_image,
                              const thrust::device_vector<MagnitudeType>& d_magnitude,
                              thrust::device_vector<BlurredType>& d_blurred, thrust::device_vector<RTVType>& d_rtv,
                              const int ksize) {
        const dim3 grid_dim{static_cast<std::uint32_t>(height_)};
        const dim3 block_dim{static_cast<std::uint32_t>(width_)};
        compute_blur_and_rtv_kernel<<<grid_dim, block_dim>>>(
            d_image.data().get(), d_magnitude.data().get(), d_blurred.data().get(), d_rtv.data().get(), ksize,
            width_, height_);
        CUDASafeCall();
    }

    template <typename BlurredType, typename RTVType, typename GuideType>
    void compute_guide(const thrust::device_vector<BlurredType>& d_blurred, const thrust::device_vector<RTVType>& d_rtv,
                       thrust::device_vector<GuideType>& d_guide, const int ksize) {
        const dim3 grid_dim{static_cast<std::uint32_t>(height_)};
        const dim3 block_dim{static_cast<std::uint32_t>(width_)};
        compute_guide_kernel<<<grid_dim, block_dim>>>(
            d_blurred.data().get(), d_rtv.data().get(), d_guide.data().get(), ksize, width_, height_);
        CUDASafeCall();
    }

    template <typename ImageType, typename GuideType>
    void joint_bilateral_filter(const thrust::device_vector<ImageType>& d_src,
                                const thrust::device_vector<GuideType>& d_guide,
                                thrust::device_vector<ImageType>& d_dst,
                                const int ksize, const float sigma_space, const float sigma_color) {
        const auto khalf  = ksize / 2;

        thrust::host_vector<float> h_kernel_space(ksize * ksize);
        for (int ky = -khalf; ky <= khalf; ky++) {
            for (int kx = -khalf; kx <= khalf; kx++) {
                const auto kidx = (ky + khalf) * ksize + (kx + khalf);
                h_kernel_space[kidx] = std::exp(-(kx * kx + ky * ky) / (2 * sigma_space * sigma_space));
            }
        }
        thrust::device_vector<float> kernel_space = h_kernel_space;

        thrust::host_vector<float> h_kernel_color_table(255 * 255);
        for (int i = 0; i < h_kernel_color_table.size(); i++) {
            h_kernel_color_table[i] = std::exp(-i / (2 * sigma_color * sigma_color));
        }
        thrust::device_vector<float> kernel_color_table = h_kernel_color_table;

        const dim3 grid_dim{static_cast<std::uint32_t>(height_)};
        const dim3 block_dim{static_cast<std::uint32_t>(width_)};
        joint_bilateral_filter_kernel<<<grid_dim, block_dim>>>(
            d_src.data().get(), d_guide.data().get(), d_dst.data().get(), ksize, kernel_space.data().get(),
            kernel_color_table.data().get(), width_, height_);
        CUDASafeCall();
    }

private:
    static constexpr auto jbf_sigma_color = 0.05f * 1.73205080757f; // 0.05 * sqrt(3)

    const int width_;
    const int height_;

    thrust::device_vector<std::uint8_t> d_src_n_;
    thrust::device_vector<float>        d_blurred_;
    thrust::device_vector<float>        d_magnitude_;
    thrust::device_vector<float>        d_rtv_;
    thrust::device_vector<std::uint8_t> d_guide_;
};

namespace cuda {

void bilateral_texture_filter(const cv::Mat3b& src, cv::Mat3b& dst, const int ksize = 9, const int nitr = 3,
                                     const bool debug_print = false) {
    const auto width  = src.cols;
    const auto height = src.rows;

    ::thrust::device_vector<::std::uint8_t> d_src(width * height * 3);
    ::thrust::device_vector<::std::uint8_t> d_dst(width * height * 3);
    ::thrust::copy(src.ptr<::std::uint8_t>(), src.ptr<::std::uint8_t>() + width * height * 3, d_src.begin());

    CudaBilateralTextureFilterImpl impl(width, height);
    impl.execute(d_src, d_dst, ksize, nitr, debug_print);

    dst.create(src.size());
    ::thrust::copy(d_dst.begin(), d_dst.end(), dst.ptr<::std::uint8_t>());
}

} // namespace cuda
