#include "hip/hip_runtime.h"
#include "bilateral_filter_impl.cuh"
#include "device_utilities.cuh"
#include "host_utilities.hpp"

constexpr auto kernel_color_table_len = 256 * 3;

__global__ void bilateral_filter_kernel(
    const std::uint8_t* const src,
    std::uint8_t* const       dst,
    const int                 ksize,
    const float* const        kernel_space,
    const float* const        kernel_color_table,
    const int                 width,
    const int                 height
) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int stride_3ch = width * 3;
    const int radius  = ksize / 2;

    extern __shared__ float s_kernel_buffer[];
    auto s_kernel_space       = &s_kernel_buffer[0];
    auto s_kernel_color_table = &s_kernel_buffer[ksize * ksize];

    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < ksize * ksize; i += blockDim.x * blockDim.y) {
        s_kernel_space[i] = kernel_space[i];
    }
    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < kernel_color_table_len; i += blockDim.x * blockDim.y) {
        s_kernel_color_table[i] = kernel_color_table[i];
    }
    __syncthreads();

    if (x >= width || y >= height) {
        return;
    }

    const auto get_kernel_space = [ksize, radius, s_kernel_space](const int kx, const int ky) {
        return s_kernel_space[(ky + radius) * ksize + (kx + radius)];
    };

    const auto get_kernel_color = [s_kernel_color_table](const auto a, const auto b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = abs(diff0) + abs(diff1) + abs(diff2);
        return s_kernel_color_table[color_distance];
    };

    const auto src_center_pix = src + stride_3ch * y + x * 3;
    auto sum0 = 0.f;
    auto sum1 = 0.f;
    auto sum2 = 0.f;
    auto sumk = 0.f;

    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);
            const auto src_pix   = src + stride_3ch * y_clamped + x_clamped * 3;
            const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(src_center_pix, src_pix);

            sum0 += src_pix[0] * kernel;
            sum1 += src_pix[1] * kernel;
            sum2 += src_pix[2] * kernel;
            sumk += kernel;
        }
    }

    dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk);
    dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk);
    dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk);
}

__global__ void joint_bilateral_filter_kernel(
    const std::uint8_t* const src,
    const std::uint8_t* const guide,
    std::uint8_t* const       dst,
    const int                 ksize,
    const float* const        kernel_space,
    const float* const        kernel_color_table,
    const int                 width,
    const int                 height
) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int stride_3ch = width * 3;
    const int radius  = ksize / 2;

    extern __shared__ float s_kernel_buffer[];
    auto s_kernel_space       = &s_kernel_buffer[0];
    auto s_kernel_color_table = &s_kernel_buffer[ksize * ksize];

    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < ksize * ksize; i += blockDim.x * blockDim.y) {
        s_kernel_space[i] = kernel_space[i];
    }
    for (int i = blockDim.x * threadIdx.y + threadIdx.x; i < kernel_color_table_len; i += blockDim.x * blockDim.y) {
        s_kernel_color_table[i] = kernel_color_table[i];
    }
    __syncthreads();

    if (x >= width || y >= height) {
        return;
    }

    const auto get_kernel_space = [ksize, radius, s_kernel_space](const int kx, const int ky) {
        return s_kernel_space[(ky + radius) * ksize + (kx + radius)];
    };

    const auto get_kernel_color = [s_kernel_color_table](const auto a, const auto b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = abs(diff0) + abs(diff1) + abs(diff2);
        return s_kernel_color_table[color_distance];
    };

    const auto guide_center_pix = guide + stride_3ch * y + x * 3;
    auto sum0 = 0.f;
    auto sum1 = 0.f;
    auto sum2 = 0.f;
    auto sumk = 0.f;

    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto x_clamped = clamp(x + kx, 0, width - 1);
            const auto y_clamped = clamp(y + ky, 0, height - 1);
            const auto pix       = src + stride_3ch * y_clamped + x_clamped * 3;
            const auto guide_pix = guide + stride_3ch * y_clamped + x_clamped * 3;
            const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(guide_center_pix, guide_pix);

            sum0 += pix[0] * kernel;
            sum1 += pix[1] * kernel;
            sum2 += pix[2] * kernel;
            sumk += kernel;
        }
    }

    dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk);
    dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk);
    dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk);
}

CudaBilateralFilter::Impl::Impl(
    const int   width,
    const int   height,
    const int   ksize,
    const float sigma_space,
    const float sigma_color)
: width_(width),
  height_(height),
  ksize_(ksize),
  sigma_space_(sigma_space),
  sigma_color_(sigma_color),
  d_kernel_space_(ksize * ksize),
  d_kernel_color_table_(kernel_color_table_len) {
    const auto gauss_color_coeff = -1.f / (2 * sigma_color * sigma_color);
    const auto gauss_space_coeff = -1.f / (2 * sigma_space * sigma_space);
    const auto radius  = ksize / 2;

    std::vector<float> h_kernel_space(ksize_ * ksize_);
    for (int ky = -radius; ky <= radius; ky++) {
        for (int kx = -radius; kx <= radius; kx++) {
            const auto kidx = (ky + radius) * ksize_ + (kx + radius);
            const auto r2 = kx * kx + ky * ky;
            if (r2 > radius * radius) {
                continue;
            }
            h_kernel_space[kidx] = std::exp(r2 * gauss_space_coeff);
        }
    }
    thrust::copy(h_kernel_space.begin(), h_kernel_space.end(), d_kernel_space_.begin());

    std::vector<float> h_kernel_color_table(kernel_color_table_len);
    for (int i = 0; i < h_kernel_color_table.size(); i++) {
        h_kernel_color_table[i] = std::exp((i * i) * gauss_color_coeff);
    }
    thrust::copy(h_kernel_color_table.begin(), h_kernel_color_table.end(), d_kernel_color_table_.begin());
}

void CudaBilateralFilter::Impl::bilateral_filter(
    const std::uint8_t* const d_src,
    std::uint8_t* const       d_dst
) const {
    const std::uint32_t block_width  = 32u;
    const std::uint32_t block_height = 32u;
    const std::uint32_t grid_width   = (width_  + block_width  - 1) / block_width;
    const std::uint32_t grid_height  = (height_ + block_height - 1) / block_height;

    const dim3 grid_dim (grid_width, grid_height);
    const dim3 block_dim(block_width, block_height);
    const std::uint32_t smem_size = (d_kernel_space_.size() + d_kernel_color_table_.size()) * sizeof(float);
    bilateral_filter_kernel<<<grid_dim, block_dim, smem_size>>>(
        d_src, d_dst, ksize_, d_kernel_space_.data().get(), d_kernel_color_table_.data().get(), width_, height_);
    CUDASafeCall();
}

void CudaBilateralFilter::Impl::joint_bilateral_filter(
    const std::uint8_t* const d_src,
    const std::uint8_t* const d_guide,
    std::uint8_t* const       d_dst
) const {
    const std::uint32_t block_width  = 32u;
    const std::uint32_t block_height = 32u;
    const std::uint32_t grid_width   = (width_  + block_width  - 1) / block_width;
    const std::uint32_t grid_height  = (height_ + block_height - 1) / block_height;

    const dim3 grid_dim (grid_width, grid_height);
    const dim3 block_dim(block_width, block_height);
    const std::uint32_t smem_size = (d_kernel_space_.size() + d_kernel_color_table_.size()) * sizeof(float);
    joint_bilateral_filter_kernel<<<grid_dim, block_dim, smem_size>>>(
        d_src, d_guide, d_dst, ksize_, d_kernel_space_.data().get(), d_kernel_color_table_.data().get(),
        width_, height_);
    CUDASafeCall();
}

CudaBilateralFilter::CudaBilateralFilter(
    const int   width,
    const int   height,
    const int   ksize,
    const float sigma_space,
    const float sigma_color
) {
    impl_ = new CudaBilateralFilter::Impl(width, height, ksize, sigma_space, sigma_color);
}

CudaBilateralFilter::~CudaBilateralFilter() {
    delete impl_;
}

void CudaBilateralFilter::bilateral_filter(
    const std::uint8_t* const d_src,
    std::uint8_t* const d_dst
) const {
    impl_->bilateral_filter(d_src, d_dst);
    hipDeviceSynchronize();
}


void CudaBilateralFilter::joint_bilateral_filter(
    const std::uint8_t* const d_src,
    const std::uint8_t* const d_guide,
    std::uint8_t* const d_dst
) const {
    impl_->joint_bilateral_filter(d_src, d_guide, d_dst);
    hipDeviceSynchronize();
}
