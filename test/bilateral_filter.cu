#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include "random_array.hpp"

#include "bilateral_filter_impl.cuh"

class RefBilateralFilterImpl {
private:
    auto get_kernel_space(const int kx, const int ky) {
        const int radius = ksize_ / 2;
        return kernel_space_[(ky + radius) * ksize_ + (kx + radius)];
    }

    auto get_kernel_color(const std::uint8_t* const a, const std::uint8_t* const b) {
        const auto diff0 = static_cast<int>(a[0]) - static_cast<int>(b[0]);
        const auto diff1 = static_cast<int>(a[1]) - static_cast<int>(b[1]);
        const auto diff2 = static_cast<int>(a[2]) - static_cast<int>(b[2]);
        const auto color_distance = std::abs(diff0) + std::abs(diff1) + std::abs(diff2);
        return kernel_color_table_[color_distance];
    }

public:
    RefBilateralFilterImpl(
        const int width,
        const int height,
        const int ksize = 9,
        const float sigma_space = 10.f,
        const float sigma_color = 30.f)
    : width_(width),
      height_(height),
      ksize_(ksize),
      kernel_space_(new float[ksize_ * ksize_]),
      kernel_color_table_(new float[256 * 3]) {
        const auto gauss_color_coeff = -1.f / (2 * sigma_color * sigma_color);
        const auto gauss_space_coeff = -1.f / (2 * sigma_space * sigma_space);
        const auto radius  = ksize_ / 2;

        for (int ky = -radius; ky <= radius; ky++) {
            for (int kx = -radius; kx <= radius; kx++) {
                const auto kidx = (ky + radius) * ksize_ + (kx + radius);
                const auto r2 = kx * kx + ky * ky;
                if (r2 > radius * radius) {
                    kernel_space_[kidx] = 0.f;
                    continue;
                }
                kernel_space_[kidx] = std::exp(r2 * gauss_space_coeff);
            }
        }

        for (int i = 0; i < 256 * 3; i++) {
            kernel_color_table_[i] = std::exp((i * i) * gauss_color_coeff);
        }
    }

    void bilateral_filter(
        const std::uint8_t* const src,
        std::uint8_t* const dst
    ) {
        const auto stride_3ch = width_ * 3;
        const auto radius = ksize_ / 2;

        for (int y = 0; y < height_; y++) {
            for (int x = 0; x < width_; x++) {
                const auto src_center_pix = src + stride_3ch * y + x * 3;
                auto sum0 = 0.f;
                auto sum1 = 0.f;
                auto sum2 = 0.f;
                auto sumk = 0.f;

                for (int ky = -radius; ky <= radius; ky++) {
                    for (int kx = -radius; kx <= radius; kx++) {
                        const auto x_clamped = std::clamp(x + kx, 0, width_ - 1);
                        const auto y_clamped = std::clamp(y + ky, 0, height_ - 1);
                        const auto src_pix   = src + stride_3ch * y_clamped + x_clamped * 3;
                        const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(src_center_pix, src_pix);

                        sum0 += src_pix[0] * kernel;
                        sum1 += src_pix[1] * kernel;
                        sum2 += src_pix[2] * kernel;
                        sumk += kernel;
                    }
                }

                dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk);
                dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk);
                dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk);
            }
        }
    }

    void joint_bilateral_filter(
        const std::uint8_t* const src,
        const std::uint8_t* const guide,
        std::uint8_t* const dst
    ) {
        const auto stride_3ch = width_ * 3;
        const auto radius = ksize_ / 2;

        for (int y = 0; y < height_; y++) {
            for (int x = 0; x < width_; x++) {
                const auto guide_center_pix = guide + stride_3ch * y + x * 3;
                auto sum0 = 0.f;
                auto sum1 = 0.f;
                auto sum2 = 0.f;
                auto sumk = 0.f;

                for (int ky = -radius; ky <= radius; ky++) {
                    for (int kx = -radius; kx <= radius; kx++) {
                        const auto x_clamped = std::clamp(x + kx, 0, width_ - 1);
                        const auto y_clamped = std::clamp(y + ky, 0, height_ - 1);
                        const auto src_pix   = src + stride_3ch * y_clamped + x_clamped * 3;
                        const auto guide_pix = guide + stride_3ch * y_clamped + x_clamped * 3;
                        const auto kernel    = get_kernel_space(kx, ky) * get_kernel_color(guide_center_pix, guide_pix);

                        sum0 += src_pix[0] * kernel;
                        sum1 += src_pix[1] * kernel;
                        sum2 += src_pix[2] * kernel;
                        sumk += kernel;
                    }
                }

                dst[stride_3ch * y + x * 3 + 0] = static_cast<std::uint8_t>(sum0 / sumk);
                dst[stride_3ch * y + x * 3 + 1] = static_cast<std::uint8_t>(sum1 / sumk);
                dst[stride_3ch * y + x * 3 + 2] = static_cast<std::uint8_t>(sum2 / sumk);
            }
        }
    }

private:
    const int width_;
    const int height_;
    const int ksize_;

    std::unique_ptr<float[]> kernel_space_;
    std::unique_ptr<float[]> kernel_color_table_;
};

class CudaBilateralFilterImpl : public CudaBilateralFilter {
public:
    CudaBilateralFilterImpl(
        const int width,
        const int height,
        const int ksize = 9,
        const float sigma_space = 10.f,
        const float sigma_color = 30.f)
    : CudaBilateralFilter(width, height, ksize, sigma_space, sigma_color) {}

    void bilateral_filter(
        const std::uint8_t* const d_src,
        std::uint8_t* const d_dst
    ) {
        impl_->bilateral_filter(d_src, d_dst);
    }

    void joint_bilateral_filter(
        const std::uint8_t* const d_src,
        const std::uint8_t* const d_guide,
        std::uint8_t* const d_dst
    ) {
        impl_->joint_bilateral_filter(d_src, d_guide, d_dst);
    }
};

TEST(BilateralFilterTest, BilateralFilter) {
    constexpr auto width  = 50;
    constexpr auto height = 50;
    constexpr auto len    = width * height * 3;

    const auto src      = random_array<std::uint8_t>(len);
    const auto actual   = std::make_unique<std::uint8_t[]>(len);
    const auto expected = std::make_unique<std::uint8_t[]>(len);
    auto d_src    = thrust::device_vector<std::uint8_t>(len);
    auto d_actual = thrust::device_vector<std::uint8_t>(len);
    thrust::copy(src.get(), src.get() + len, d_src.begin());

    CudaBilateralFilterImpl cuda_impl(width, height);
    cuda_impl.bilateral_filter(d_src.data().get(), d_actual.data().get());
    thrust::copy(d_actual.begin(), d_actual.end(), actual.get());

    RefBilateralFilterImpl ref_impl(width, height);
    ref_impl.bilateral_filter(src.get(), expected.get());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            const auto actual_ptr   = &actual[width * 3 * y + x * 3];
            const auto expected_ptr = &expected[width * 3 * y + x * 3];
            EXPECT_EQ(actual_ptr[0], expected_ptr[0]) << "(x, y, ch) = (" << x << ", " << y << ", " << 0 << ")";
            EXPECT_EQ(actual_ptr[1], expected_ptr[1]) << "(x, y, ch) = (" << x << ", " << y << ", " << 1 << ")";
            EXPECT_EQ(actual_ptr[2], expected_ptr[2]) << "(x, y, ch) = (" << x << ", " << y << ", " << 2 << ")";
        }
    }
}

TEST(BilateralFilterTest, JointBilateralFilter) {
    constexpr auto width  = 50;
    constexpr auto height = 50;
    constexpr auto len    = width * height * 3;

    const auto src      = random_array<std::uint8_t>(len);
    const auto guide    = random_array<std::uint8_t>(len);
    const auto actual   = std::make_unique<std::uint8_t[]>(len);
    const auto expected = std::make_unique<std::uint8_t[]>(len);
    auto d_src    = thrust::device_vector<std::uint8_t>(len);
    auto d_guide  = thrust::device_vector<std::uint8_t>(len);
    auto d_actual = thrust::device_vector<std::uint8_t>(len);
    thrust::copy(src.get(), src.get() + len, d_src.begin());
    thrust::copy(guide.get(), guide.get() + len, d_guide.begin());

    CudaBilateralFilterImpl cuda_impl(width, height);
    cuda_impl.joint_bilateral_filter(d_src.data().get(), d_guide.data().get(), d_actual.data().get());
    thrust::copy(d_actual.begin(), d_actual.end(), actual.get());

    RefBilateralFilterImpl ref_impl(width, height);
    ref_impl.joint_bilateral_filter(src.get(), guide.get(), expected.get());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            const auto actual_ptr   = &actual[width * 3 * y + x * 3];
            const auto expected_ptr = &expected[width * 3 * y + x * 3];
            EXPECT_EQ(actual_ptr[0], expected_ptr[0]) << "(x, y, ch) = (" << x << ", " << y << ", " << 0 << ")";
            EXPECT_EQ(actual_ptr[1], expected_ptr[1]) << "(x, y, ch) = (" << x << ", " << y << ", " << 1 << ")";
            EXPECT_EQ(actual_ptr[2], expected_ptr[2]) << "(x, y, ch) = (" << x << ", " << y << ", " << 2 << ")";
        }
    }
}
